#include "hip/hip_runtime.h"
#include "GaussianFilter.cuh"
#include "SM_Loader.cuh"

void calc_filter(Kernel_Gaussian_weights &kernel, unsigned char kernel_size, double sigma);
__global__ void gaussian_kernel(unsigned char* img_device, unsigned char* out_device, unsigned char kernel_size, unsigned char radius, unsigned int sizeSM, short rows, short cols);


__constant__ Kernel_Gaussian_weights const_gw;


void GaussianFilterGPU(unsigned char* img_host,
                       unsigned char* out,
                       unsigned char kernel_size,
                       double sigma,
                       short rows,
                       short cols){

    unsigned char *out_device, *img_device;
    unsigned char radius;
    unsigned int size=rows*cols*sizeof(unsigned char);
    int sizeSMbyte;
    dim3 num_blocks, num_threads_per_block;

  
    unsigned int factor=16;
    num_threads_per_block.y=factor;
    num_threads_per_block.x=factor;
    
    num_blocks.y = rows/num_threads_per_block.y+((rows%num_threads_per_block.y)==0? 0:1);
    num_blocks.x = cols/num_threads_per_block.x+((cols%num_threads_per_block.x)==0? 0:1);

    
    Kernel_Gaussian_weights kernel;
   
    calc_filter(kernel, kernel_size, sigma);
  
    hipMemcpyToSymbol(HIP_SYMBOL(const_gw), &kernel, sizeof(kernel));

    
    hipMalloc((void**)&img_device, size);
    hipMalloc((void**)&out_device, size);

    
    hipMemcpy(img_device, img_host, size, hipMemcpyHostToDevice);

    
    sizeSMbyte = (num_threads_per_block.y+kernel_size-1)*(num_threads_per_block.x+kernel_size-1)*sizeof(unsigned char);
    unsigned int sizeSM = (num_threads_per_block.x+kernel_size-1);

   
    radius=int(floor((kernel_size-1)/2));

   
    gaussian_kernel<<<num_blocks, num_threads_per_block, sizeSMbyte>>>(img_device, out_device, kernel_size, radius, sizeSM, rows, cols);
    hipDeviceSynchronize();

    
    hipMemcpy(out, out_device, size, hipMemcpyDeviceToHost);

    
    hipFree(img_device);
    hipFree(out_device);

}


__global__ void gaussian_kernel(unsigned char* img_device,
                                unsigned char* out_device,
                                unsigned char kernel_size,
                                unsigned char radius,
                                unsigned int sizeSM,
                                short rows,
                                short cols){

    
    extern __shared__ unsigned char sm_gaussian[];

    
    SM_data_loader(sm_gaussian, img_device, sizeSM, radius, rows, cols);

    
    __syncthreads();

  

 
    float sum=0;
    for (int y=0; y<kernel_size; y++)
        for (int x=0; x<kernel_size; x++)
            sum += sm_gaussian[(threadIdx.y+y)*sizeSM+(threadIdx.x+x)]*const_gw.weights[y][x];

    
    unsigned int y = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int x = blockIdx.x*blockDim.y+threadIdx.x;

    
    if (y<rows && x<cols)
          out_device[y*cols+x] = (unsigned char)(floor(sum));

}


void calc_filter(Kernel_Gaussian_weights &kernel, unsigned char kernel_size, double sigma){

    memset(&kernel, 0, sizeof(kernel));
    double r, s = 2.0*sigma*sigma;
    double sum = 0.0;
    unsigned char radius = floor(kernel_size/2);

    
    for (int x=-radius; x<=radius; x++)
      for(int y=-radius; y<=radius; y++){
              r = sqrt(x*x + y*y);
              kernel.weights[x+radius][y+radius] = (exp(-(r*r)/s))/(M_PI*s);
              sum += kernel.weights[x+radius][y+radius];
          }

    
    for(int i=0; i<kernel_size; ++i)
        for(int j=0; j<kernel_size; ++j)
            kernel.weights[i][j] /= sum;

}
